#include "hip/hip_runtime.h"
//compile nvcc -arch=sm_35 -o CUDA1.1PaStA CUDA1.1PaStA.cu

#include <stdio.h>
#include <algorithm>    

#include <limits.h>
//#include "util.h"


// cuda error function
#define checkCuda(err) (CheckCuda(err, __FILE__, __LINE__))
static void CheckCuda(hipError_t result, const char *file, int line)
{

	if (result != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(result),
			file, line);
		exit(EXIT_FAILURE);
	}
}

// CUDA kernel. Each thread takes care of one element of c
// This function is based off of the NVIDIA code shown in class but it changed
// to reflect a min reduction
__global__ void reduceSum(float * g_idata, float * g_odata, int n, int vectLen)
{
	extern __shared__ float sdata[];

	//global index
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	//thread index
	int tid_x = threadIdx.x;
	int tid_y = threadIdx.y;
  int blockSize = blockDim.x*blockDim.y;

	//int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int globalId = i*n + j;
	int sharedIdx = tid_y*blockDim.x + tid_x;

	sdata[sharedIdx] = ((globalId < vectLen) && (i < n) && (j < n)) ? g_idata[globalId] : 0;
  __syncthreads();

	int s;
	for (s = (blockSize) / 2; s > 0; s >>= 1) {
		if (sharedIdx < s) {
			sdata[sharedIdx] += sdata[sharedIdx + s];
		}
		__syncthreads();

	}
	float sum;
	if (sharedIdx == 0) {
		sum = sdata[sharedIdx];
		atomicAdd(g_odata, sum);

	}}

__global__ void deltaReduceSum(float * g_idata, float * g_odata, int n, int vectLen, float *mean)
{
	extern __shared__ float sdata[];

	//global index
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	//thread index
	int tid_x = threadIdx.x;
	int tid_y = threadIdx.y;

	//int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int globalId = i*n + j;
	int sharedIdx = tid_y*blockDim.x + tid_x;
  int blockSize = blockDim.x*blockDim.y;
  
	sdata[sharedIdx] = ((globalId < vectLen) && (i < n) && (j < n))
		? (g_idata[globalId] - mean[0])*(g_idata[globalId] - mean[0]) : 0;
	__syncthreads();

	int s;
	for (s = (blockSize) / 2; s > 0; s >>= 1) {
		if (sharedIdx < s) {
			sdata[sharedIdx] += sdata[sharedIdx + s];
		}
		__syncthreads();

	}
	float sum;
	if (sharedIdx == 0) {
		sum = sdata[sharedIdx];
		atomicAdd(g_odata, sum);

	}
}

__global__ void histReduce(float * g_idata, float * g_odata, int n, int vectLen, int bins, float mean, float std, int offset)
{
	extern __shared__ float sdata2[];

	//global index
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	//thread index
	int tid_x = threadIdx.x;
	int tid_y = threadIdx.y;
  int blockSize = blockDim.x*blockDim.y;
  
 	int globalId = i*n + j;
	int sharedIdx = tid_y*blockDim.x + tid_x;
  int targetBin;
   if((globalId < vectLen) && (i < n) && (j < n)) {
  		if (g_idata[globalId] < mean)
  		{
        
  			targetBin = offset - (int)floor((mean - g_idata[globalId]) / std);
  		}
  		else
  		{
  			targetBin = (int)ceil((g_idata[globalId] - mean) / std) + offset;
  		}
    }
int bin;
for(bin = 0; bin < bins; bin++) {

	//int blockId = blockIdx.x + blockIdx.y * gridDim.x;
   if((globalId < vectLen) && (i < n) && (j < n)) {
		sdata2[sharedIdx] = (targetBin == bin) ? 1 : 0;
  } else {
    sdata2[sharedIdx] = 0;
  }
	__syncthreads();

	int s;
	for (s = (blockSize) / 2; s > 0; s >>= 1) {
		if (sharedIdx < s) {
			sdata2[sharedIdx] += sdata2[sharedIdx + s];
		}
		__syncthreads();

	}
	float sum;
	if (sharedIdx == 0) {
		sum = sdata2[sharedIdx];
		atomicAdd(&(g_odata[bin]), sum);

	}
 }
}


double runTest(float *v, int setSize, float min, float max, int suppressOutput);

int main()
{

	size_t setSize = 1000000000;
	//PutSeed(10);
	float * v;
	v = (float*)malloc(setSize*sizeof(float));
	float min = 1000000000000000;
	float max = -1000000000000000;

	int i;
	for (i = 0; i < setSize; i++)
	{
		v[i] = rand() / (float)RAND_MAX + rand() / (float)RAND_MAX + rand() / (float)RAND_MAX + 1;
   
		if (v[i] < min) min = v[i];
		if (v[i] > max) max = v[i];
	}

	double time;
	//time = runTest(v, 10000, min, max, 0);
	
	int m, run;
	for (m = 6; m < 10; m++) {
	for (run = 0; run < 11; run++){
	time = runTest(v, pow(10,m), min, max, 1);
 
  if(run >= 5)	printf("CUDA,%d,%f\n", m, time);
	}
	}
}
double runTest(float *h_v, int setSize, float min, float max, int suppressOutput) {

	hipEvent_t start = 0;
	hipEvent_t stop = 0;
	float time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float *h_sum;

	//device vectors
	float *d_v;
	float *d_sum;

	size_t bytes = setSize*sizeof(float);
	int gridSide = ceil(sqrt(setSize) / 32.0);

	size_t outbytes = sizeof(float);

	// Number of threads in each thread block
	h_sum = (float*)malloc(outbytes);
	h_sum[0] = 0;
	int i;


	checkCuda(hipMalloc((void**)&d_v, bytes));
	checkCuda(hipMalloc((void**)&d_sum, outbytes));


	checkCuda(hipMemcpy(d_v, h_v, bytes, hipMemcpyHostToDevice));


	int blockSide = 32;
	hipEventRecord(start, 0);
	dim3 block(blockSide, blockSide);
	dim3 grid(gridSide, gridSide);

	reduceSum << <grid, block, blockSide * blockSide * sizeof(float) >> >(d_v, d_sum, ceil(sqrt(setSize)), setSize);


	checkCuda(hipMemcpy(h_sum, d_sum, outbytes, hipMemcpyDeviceToHost));

	



	float * h_mean;
	float * h_delta;

	h_mean = (float*)malloc(outbytes);
	h_delta = (float*)malloc(outbytes);

	float * d_mean;
	float * d_delta;

	checkCuda(hipMalloc((void**)&d_mean, outbytes));
	checkCuda(hipMalloc((void**)&d_delta, outbytes));

	h_mean[0] = h_sum[0] / setSize;

	checkCuda(hipMemcpy(d_mean, h_mean, outbytes, hipMemcpyHostToDevice));

	deltaReduceSum <<<grid, block, blockSide * blockSide * sizeof(float) >>>(d_v, d_delta, ceil(sqrt(setSize)), setSize, d_mean);

	checkCuda(hipMemcpy(h_delta, d_delta, outbytes, hipMemcpyDeviceToHost));

	float h_std;
	
	h_std = sqrt(h_delta[0] / ((float)setSize - 1.0));


	int h_bins = ceil((max - min) / h_std) + 1; 
 

  
	float * h_histogram;
  float * d_histogram;
  h_histogram = (float*)malloc(sizeof(float)*h_bins);
  checkCuda(hipMalloc((void**)&d_histogram, sizeof(float)*h_bins));

	int h_offset = floor((h_mean[0] - min) / h_std);
    
  histReduce <<<grid, block, blockSide * blockSide * sizeof(float) >>>(d_v, d_histogram, ceil(sqrt(setSize)), setSize,h_bins, h_mean[0], h_std, h_offset);
  checkCuda(hipMemcpy(h_histogram, d_histogram, sizeof(float)*h_bins, hipMemcpyDeviceToHost));
/*	for (i = 0; i < setSize; i++) {
		if (h_v[i] < h_mean[0])
		{
			histogram[offset - (int)floor((h_mean[0] - h_v[i]) / h_std[0])]++;
		}
		else
		{
			histogram[(int)ceil((h_v[i] - h_mean[0]) / h_std[0]) + offset]++;
		}
	}*/

	if (suppressOutput == 0) {
		printf("(sum %f)\n", h_sum[0]);
		printf("Here are the results:\n");
		printf("Mean: %f\nStandard Deviation: %f\n", h_mean[0], h_std);

		for (i = 0; i < h_bins; i++) {
			if (i <= h_offset) {
				printf("%d stds: %f\n", (i - h_offset - 1), h_histogram[i] / (float)setSize);
			}
			else
			{
				printf("%d stds: %f\n", i - h_offset, h_histogram[i] / (float)setSize);
			}
		}
	}





	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Release device memory
	hipFree(d_v);
	hipFree(d_sum);
	hipFree(d_mean);
	hipFree(d_delta);
	hipEventElapsedTime(&time, start, stop);

	//	printf("Real min: %.20f, myMin: %.20f\n", realMin, myMin);
	if(suppressOutput == 0) printf("Time for the kernel: %f\n", time);

	return time/1000.0;
}